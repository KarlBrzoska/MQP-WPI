
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS 2048
#define NUM_SAMPLES 1000000000000

__global__ void initRandomStates(hiprandState *states, unsigned long long seed) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void monteCarloPiKernel(unsigned long long *count, hiprandState *states) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned long long local_count = 0;
    hiprandState localState = states[idx];

    // Each thread will do an equal amount of work
    unsigned long long iterations = NUM_SAMPLES / (blockDim.x * gridDim.x);

    for (unsigned long long i = 0; i < iterations; ++i) {
        float x = hiprand_uniform(&localState);
        float y = hiprand_uniform(&localState);
        if (x * x + y * y <= 1.0f) {
            local_count++;
        }
    }

    // Copy state back to global memory
    states[idx] = localState;

    // Use atomicAdd to avoid race condition
    atomicAdd(count, local_count);
}

int main() {
    // Start CPU timing
    clock_t start_cpu = clock();

    unsigned long long *d_count;
    hiprandState *d_states;

    // Allocate memory on the device
    hipMalloc(&d_count, sizeof(unsigned long long));
    hipMalloc(&d_states, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(hiprandState));

    // Initialize d_count to 0
    hipMemset(d_count, 0, sizeof(unsigned long long));

    // Setup PRNG states
    initRandomStates<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_states, 1337ULL);

    // Wait for GPU to finish before launching the main kernel
    hipDeviceSynchronize();

    // Create CUDA events for timing
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // Record the start event for GPU timing
    hipEventRecord(start_gpu, NULL);

    // Run kernel
    monteCarloPiKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_count, d_states);

    // Record the stop event for GPU timing
    hipEventRecord(stop_gpu, NULL);
    hipEventSynchronize(stop_gpu);

    // Calculate the elapsed time for GPU operations
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_gpu, stop_gpu);

    // Copy count back to host
    unsigned long long h_count;
    hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Calculate pi
    double pi = 4.0 * h_count / NUM_SAMPLES;

    // End CPU timing
    clock_t end_cpu = clock();
    double cpu_time_used = ((double) (end_cpu - start_cpu)) / CLOCKS_PER_SEC;

    printf("Estimated pi: %f\n", pi);
    printf("GPU time elapsed: %f milliseconds\n", milliseconds);
    printf("Total CPU time used: %f seconds\n", cpu_time_used);

    // Cleanup
    hipFree(d_count);
    hipFree(d_states);
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    return 0;
}
